
#include <hip/hip_runtime.h>
#include <stdio.h>	// only for test routine!

__global__ void uchar4_to_float4(float4* output, uchar4* input, int width, int height)
{
	int posX = blockDim.x * blockIdx.x + threadIdx.x;
	int posY = blockDim.y * blockIdx.y + threadIdx.y;

	if(posX < width && posY < height) {
		// compute position in image
		int i = posY * width + posX;
		output[i].x = input[i].x;
		output[i].y = input[i].y;
		output[i].z = input[i].z;
		output[i].w = input[i].w;
	}
}


int convert_uchar4_to_float4(float4** d_pOutput, uchar4* d_pInput,
								int width, int height)
{
	hipError_t e;
	hipMalloc((void**)d_pOutput, sizeof(float4) * width * height);
		e = hipGetLastError();
		if(e) return -1;

	dim3 threads(16,16);
	dim3 grid( (int)ceil(width / (float)threads.x), (int)ceil(height / (float)threads.y));
	uchar4_to_float4<<< grid, threads >>>(*d_pOutput, d_pInput, width, height);
		e = hipGetLastError();
		if(e) return -2;

	return 0;
}

void test_convert_uchar4_to_float4(float4* d_pFloatImg, int width, int height)
{
	float4* h_pFloatImg = (float4*)malloc(sizeof(float4) * width * height);
	if(!h_pFloatImg) {
		printf("test_convert_float4_to_float4: malloc failed\n");
		return;
	}

	hipMemcpy(h_pFloatImg, d_pFloatImg, sizeof(float4) * width * height, hipMemcpyDeviceToHost);

	FILE* fp = fopen("float_image.txt","w");
	if(!fp) {
		printf("test_convert_float4_to_float4: failed to open file\n");
		return;
	}
	for(int y=0; y < height; y++) {
		for(int x=0; x < width; x++) {
			fprintf(fp, "(%.0f,%.0f,%.0f,%.0f)",
					h_pFloatImg[y*width + x].x,
					h_pFloatImg[y*width + x].y,
					h_pFloatImg[y*width + x].z,
					h_pFloatImg[y*width + x].w);
		}
		fprintf(fp, "\n");
	}
	fclose(fp);
}
