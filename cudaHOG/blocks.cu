#include "hip/hip_runtime.h"
#include <stdio.h>

#include "global.h"

hipArray* d_pGaussWeights = NULL;
texture<float, 2, hipReadModeElementType> t_gaussian_weights;
// float4 -x UL - y UR - z BL - w BR -- one lookup table for each cell in the block
hipArray* d_pBilinearWeights = NULL;
texture<float4, 2, hipReadModeElementType> t_bilinear_weights;


__host__ int prepareGaussWeights()
{
	const float cX = HOG_BLOCK_WIDTH / 2 - 0.5f;
	const float cY = HOG_BLOCK_HEIGHT / 2 - 0.5f;
	float h_pGauss[HOG_BLOCK_WIDTH][HOG_BLOCK_HEIGHT];

	for(int y=0; y < HOG_BLOCK_HEIGHT; y++) {
		for(int x=0; x < HOG_BLOCK_WIDTH; x++) {
			h_pGauss[x][y] = 1.f /(2.f * (float)M_PI * SIGMA) * exp(- 0.5f * ( (x-cX)*(x-cX)/(SIGMA*SIGMA) + (y-cY)*(y-cY)/(SIGMA*SIGMA) ) );
		}
	}
	// normalize to 1
	float sum = 0;
	for(int x=0; x < HOG_BLOCK_WIDTH; x++) {
		for(int y=0; y < HOG_BLOCK_HEIGHT; y++) {
			sum += h_pGauss[x][y];
		}
	}
	for(int x=0; x < HOG_BLOCK_WIDTH; x++) {
		for(int y=0; y < HOG_BLOCK_HEIGHT; y++) {
			h_pGauss[x][y] /= sum;
		}
	}

#ifdef DEBUG_GAUSS_WEIGHTS
	// output gauss weights to file for debugging
	FILE* fp = fopen("gauss_weights.txt", "w");
	for(int x=0; x < 16; x++) {
		for(int y=0; y < 16; y++)
			fprintf(fp, "%.8f ", h_pGauss[x][y] );
		fprintf(fp, "\n");
	}
	fclose(fp);
#endif

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	hipMallocArray( &d_pGaussWeights, &channelDesc, HOG_BLOCK_WIDTH, HOG_BLOCK_HEIGHT);
		ONFAIL("malloc array\n");

	hipMemcpyToArray(d_pGaussWeights, 0, 0, h_pGauss,
						HOG_BLOCK_WIDTH * HOG_BLOCK_HEIGHT * sizeof(float),
						hipMemcpyHostToDevice);
		ONFAIL("memcpy to array\n");

	hipBindTextureToArray( t_gaussian_weights, d_pGaussWeights, channelDesc);
		ONFAIL("bind tex to array\n");

	return 0;
}

#ifdef DEBUG_GAUSS_WEIGHTS
__global__ void testGaussWeights(float* d_pOutput)
{
	if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0 ) {
		for(int blockX=0; blockX < HOG_BLOCK_WIDTH; blockX++) {
			for(int blockY=0; blockY < 16; blockY+=8) {
				for(int i=0; i < HOG_CELL_SIZE; i++) {
					float g = tex2D( t_gaussian_weights, blockX, blockY+i);
					d_pOutput[HOG_BLOCK_WIDTH*(blockY+i)+blockX] = g;
				}
			}
		}
	}

}
#endif

__host__ int prepareBilinearWeights()
{
	float* h_pWeights = (float*)malloc(sizeof(float) * 4 * 2 * HOG_CELL_SIZE * 2 * HOG_CELL_SIZE);
	if(!h_pWeights) {
		printf("prepareBilinearWeights: malloc failed!\n");
		return -1;
	}

	float h_weights_L[HOG_CELL_SIZE*2];	// left cells
	float h_weights_R[HOG_CELL_SIZE*2];	// right cells (left mirrored)
	float h_weights_T[HOG_CELL_SIZE*2];	// upper cells
	float h_weights_B[HOG_CELL_SIZE*2];	// bottom cells

	memset(h_weights_L, 0, sizeof(float) * HOG_CELL_SIZE * 2);
	memset(h_weights_R, 0, sizeof(float) * HOG_CELL_SIZE * 2);
	memset(h_weights_T, 0, sizeof(float) * HOG_CELL_SIZE * 2);
	memset(h_weights_B, 0, sizeof(float) * HOG_CELL_SIZE * 2);

	int d = 9;
	int g = 0;
	for(int x=0; x < 4; x++, g+=2) {
			h_weights_L[x] = (d+g) / 16.f;
			h_weights_T[x] = (d+g) / 16.f;
	}
	d = 15;
	g = 0;
	for(int x=4; x < 12; x++, g+=2) {
		h_weights_L[x] = (d-g) / 16.f;
		h_weights_T[x] = (d-g) / 16.f;
	}

	for(int x=0; x < HOG_CELL_SIZE*2; x++) {
		h_weights_R[x] = h_weights_L[HOG_CELL_SIZE*2-1-x];
		h_weights_B[x] = h_weights_T[HOG_CELL_SIZE*2-1-x];
	}

	// prepare a complete lookup table for each pixel in a _block_ !
	for(int x=0; x < 2*HOG_CELL_SIZE; x++) {
		for(int y=0; y < 2*HOG_CELL_SIZE; y++) {
			const int idx = 4 * (y * 2*HOG_CELL_SIZE + x);
			// float4 -- x UL - y UR - z BL - w BR
			h_pWeights[idx+0] = h_weights_L[x] * h_weights_T[y];
			h_pWeights[idx+1] = h_weights_R[x] * h_weights_T[y];
			h_pWeights[idx+2] = h_weights_L[x] * h_weights_B[y];
			h_pWeights[idx+3] = h_weights_R[x] * h_weights_B[y];
		}
	}

#ifdef DEBUG_BILINEAR_WEIGHTS
	// output table to file for debugging
	int j =0;
	FILE* fp = fopen("bilweights.txt", "w");
	for(int i=0; i < 4; i++) {
		j=0;
		for(int x=0; x < 4*16*16; x+=4) {
			fprintf(fp, "%.3f ", h_pWeights[x+i] );
			j++;
			if(j == 16 ) { j=0; fprintf(fp, "\n"); }
		}
		fprintf(fp, "\n\n");
	}
	fclose(fp);
#endif

	// export the lookup table as a texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

	hipMallocArray( &d_pBilinearWeights, &channelDesc, 2*HOG_CELL_SIZE, 2*HOG_CELL_SIZE);
		ONFAIL("malloc weights array\n");

	hipMemcpyToArray(d_pBilinearWeights, 0, 0, h_pWeights,
						sizeof(float) * 4 * 2*HOG_CELL_SIZE * 2*HOG_CELL_SIZE,
						hipMemcpyHostToDevice);
		ONFAIL("memcpy to weights array\n");

	hipBindTextureToArray( t_bilinear_weights, d_pBilinearWeights, channelDesc);
		ONFAIL("bind tex to weights array\n");

	free(h_pWeights);
	return 0;
}


#ifdef DEBUG_BILINEAR_WEIGHTS
__global__ void testBilinearWeights(float* d_pOutput)
{
	if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0 ) {
		for(int i=0; i < HOG_BLOCK_HEIGHT; i++) {
			for(int j=0; j < HOG_BLOCK_WIDTH; j++) {
				float4 weights = tex2D(t_bilinear_weights, j, i);
				d_pOutput[HOG_BLOCK_WIDTH*i+j] = weights.w;
			}
		}
	}
}
#endif


__host__ int blocks_finalize()
{
	hipUnbindTexture(t_gaussian_weights);
		ONFAIL("t_gaussian_weights");
	hipFreeArray(d_pGaussWeights);
		ONFAIL("hipFree: d_pGaussianWeights");

	hipUnbindTexture(t_bilinear_weights);
		ONFAIL("t_bilinear_weights");
	hipFreeArray(d_pBilinearWeights);
		ONFAIL("hipFree: d_pBilinearWeights");

	return 0;
}


// one thread for each cell column -- 8 * 4 threads -- 32 threads
// 1. load gradients & magnitudes in shared mem (2 floats * 576 pixels)
// 2. compute cell histograms
// 3. combine cell histograms and normalize
__global__ void d_compute_blocks(int width, int height, float2* d_pGradMag, float* d_blocks)
{
	volatile __shared__ float	s_block[32][2*2][NBINS];		// 32 threads, 4 cells, 9 bins
	volatile __shared__ float	s_squares[4];

// copy relevant gradient data for the block
	const int cellIdx = threadIdx.x;	// 0-3	cells in block
	const int columnIdx = threadIdx.y;	// 0-7	columns
	const int sIdx = threadIdx.y * blockDim.x + threadIdx.x;	// which of the 32 threads are we?

	// position of the upper-most pixel in the column for this thread
	const int blockX = (cellIdx % 2)*HOG_CELL_SIZE + columnIdx;
	const int blockY = cellIdx < 2 ? 0 : HOG_CELL_SIZE;

	const int pixelX = blockIdx.x * (HOG_BLOCK_WIDTH/2) + blockX;		// we assume 50% overlap
	const int pixelY = blockIdx.y * (HOG_BLOCK_HEIGHT/2) + blockY;

	// initialize all bins for this thread
	for(int i=0; i < NBINS; i++) {
		for(int cell =0; cell < HOG_BLOCK_CELLS_X*HOG_BLOCK_CELLS_Y; cell++)
			s_block[sIdx][cell][i] = 0.f;
	}

	__syncthreads();
// ----------------------------------------------------------------------------


// for each pixel in the column of this thread
	if(pixelX < width && pixelY < height) {
	for(int i=0; i < HOG_CELL_SIZE; i++)
	{
// decide which cells to contribute to
// compute the contribution by weights
// magnitude * gaussian_weight * trilinear_interpolation
		const int pixelIdx = (pixelY + i) * width + pixelX;

		float magnitude = d_pGradMag[pixelIdx].y;
		float contribution = magnitude * tex2D(t_gaussian_weights, blockY+i, blockX);

	// calculate contribution to the two bins
		float binSize = 180.f / NBINS;

		float orientation = d_pGradMag[pixelIdx].x - binSize/2.f;
		if(orientation < 0) orientation += 180.f;
		float delta = (orientation * NBINS) / 180.f;

		int leftBin = (int)floorf( delta );
		delta -= leftBin;
		int rightBin = leftBin >= NBINS-1 ? 0 : leftBin+1;
		if( leftBin < 0 ) leftBin = NBINS -1;

		float rightContribution = contribution * (delta);
		float leftContribution = contribution * (1-delta);

	// add contributions to cells (with appropriate bilinear weights)
		float4 weights = tex2D(t_bilinear_weights, blockX, blockY+i);
		s_block[sIdx][0][leftBin] += leftContribution * weights.x;
		s_block[sIdx][0][rightBin]+= rightContribution * weights.x;

		s_block[sIdx][1][leftBin] += leftContribution * weights.y;
		s_block[sIdx][1][rightBin]+= rightContribution * weights.y;

		s_block[sIdx][2][leftBin] += leftContribution * weights.z;
		s_block[sIdx][2][rightBin]+= rightContribution * weights.z;

		s_block[sIdx][3][leftBin] += leftContribution * weights.w;
		s_block[sIdx][3][rightBin]+= rightContribution * weights.w;
	}
	}
	__syncthreads();
// ----------------------------------------------------------------------------
// reduce histograms in shared mem to one histogram
	if(threadIdx.y == 0)
	{
		// first reduce all the column results into one column
		for(int i=1; i < 32; i++) {
			for(int bin=0; bin < NBINS; bin++) {
				s_block[0][threadIdx.x][bin] += s_block[i][threadIdx.x][bin];
			}
		}
	}
	__syncthreads();
// ----------------------------------------------------------------------------
// normalize the block histogram - L2+Hys normalization

	const float epsilon = 0.036f * 0.036f;	// magic numbers
	const float eHys	= 0.1f * 0.1f;
	const float clipThreshold = 0.2f;

	if(threadIdx.y == 0 ) {
		float ls = 0.f;
		for(int j=0; j < NBINS; j++) {
			ls += s_block[0][threadIdx.x][j] * s_block[0][threadIdx.x][j];
		}
		s_squares[threadIdx.x] = ls;
	}
	__syncthreads();
	if(threadIdx.y == 0 && threadIdx.x == 0 ) {
		s_squares[0] += s_squares[1] + s_squares[2] + s_squares[3];
	}
	__syncthreads();
	// we use rsqrtf (reciprocal sqrtf) because of CUDA pecularities
	float normalization = rsqrtf(s_squares[0]+epsilon);
	// normalize and clip
	if(threadIdx.y == 0 ) {
		for(int j=0; j < NBINS; j++) {
			s_block[0][threadIdx.x][j] *= normalization;
			s_block[0][threadIdx.x][j] = s_block[0][threadIdx.x][j] > clipThreshold ? clipThreshold : s_block[0][threadIdx.x][j];
		}
	}

	// renormalize
	if(threadIdx.y == 0 ) {
		float ls = 0.f;
		for(int j=0; j < NBINS; j++) {
			ls += s_block[0][threadIdx.x][j] * s_block[0][threadIdx.x][j];
		}
		s_squares[threadIdx.x] = ls;
	}
	__syncthreads();
	if(threadIdx.y == 0 && threadIdx.x == 0 ) {
		s_squares[0] += s_squares[1] + s_squares[2] + s_squares[3];
	}

	normalization = rsqrtf(s_squares[0]+eHys);
	if(threadIdx.y == 0 ) {
		for(int j=0; j < NBINS; j++) {
			s_block[0][threadIdx.x][j] *= normalization;
		}
	}
/*	 // L1
		const float epsilon = 0.001f;
		float sum = 0.f;
		for(int i=0; i < 4; i++) {
			for(int j=0; j < NBINS; j++) {
				sum += s_block[0][i][j];
			}
		}
		float normalization = 1.f / (sum + epsilon);
		for(int i=0; i < 4; i++) {
			for(int j=0; j < NBINS; j++) {
				s_block[0][i][j] *= normalization;
			}
		}
	__syncthreads();
*/

// ----------------------------------------------------------------------------
// copy the block histogram to device mem
	if(threadIdx.y == 0 ) {
		const int writeIdx = NBINS*4 * (blockIdx.y * gridDim.x + blockIdx.x);
		for(int bin=0; bin < NBINS; bin++) {
			d_blocks[writeIdx + threadIdx.x*NBINS + bin] = s_block[0][threadIdx.x][bin];
		}
	}
}

__host__ int compute_blocks(dim3 grid, int width, int height, float2* d_pGradMag, float* d_pBlocks)
{
#ifdef DEBUG_DUMP_BLOCKS
	const int nBlocks = grid.x * grid.y;
	const int blocksMemorySize = nBlocks * HOG_BLOCK_CELLS_X * HOG_BLOCK_CELLS_Y * NBINS * sizeof(float);
	hipMemset(d_pBlocks, 0, blocksMemorySize);
#endif

	// call the cuda kernel to do the computation
	dim3 threads;
	threads.x = 4; threads.y = 8;

	d_compute_blocks<<< grid , threads >>>(width, height, d_pGradMag, d_pBlocks);
		ONFAIL("compute_blocks kernel failed");

#ifdef DEBUG_DUMP_BLOCKS
	// debug: read the result back to host memory
	float* h_pBlocks = (float*)malloc(blocksMemorySize);
	hipMemcpy(h_pBlocks, d_pBlocks, blocksMemorySize, hipMemcpyDeviceToHost);
		ONFAIL("hipMemcpy for block histograms failed"); //dirty, if we leave here..

	FILE* fp = fopen("blocks.txt", "w");
	int j=0, k=4, block_counter=0;
	fprintf(fp, "blocks: %d x %d\n", grid.x, grid.y);
	for(int x=0; x < NBINS * 4 * grid.x * grid.y; x++) {
		if(k==4) { fprintf(fp, "\n%d\n", block_counter); k=0; block_counter++; }
		fprintf(fp, "%.16f\t", h_pBlocks[x]);
		j++; if(j==NBINS){ fprintf(fp, "\n"); j=0; k++; }
	}
	fclose(fp);
	free(h_pBlocks);
#endif

	return 0;
}
